#include "hip/hip_runtime.h"
#include "cuda_tick.h"
#ifndef USE_OCTREE
#define USE_OCTREE 0
#endif
#if USE_OCTREE
#include "octree.h"
#endif

#define G 0.06

typedef struct BoundingBox
{
    vec3_m_t xmin;
    vec3_m_t ymin;
    vec3_m_t zmin;
    vec3_m_t xmax;
    vec3_m_t ymax;
    vec3_m_t zmax;
} BoundingBox;

static struct
{
    int threads_per_block_solver;
    int threads_per_block_update;
    int grid_size_solver;
    int grid_size_update;
} cuda_state;

BoundingBox compute_bounding_box(Particle *p, int n);

__global__ void newtonianSolver(Particle *particles, const double dt, int n);
__global__ void update_particle(Particle *particles, const double dt, int n);

Particle *device_particles;

void init_cuda_tick(Particle *p, int n)
{
    hipMalloc((void **)&device_particles, n * sizeof(Particle));
    hipMemcpy(device_particles, p, n * sizeof(Particle), hipMemcpyHostToDevice);
    hipOccupancyMaxPotentialBlockSize(&cuda_state.grid_size_solver, &cuda_state.threads_per_block_solver, newtonianSolver, 0, 0);
    hipOccupancyMaxPotentialBlockSize(&cuda_state.grid_size_update, &cuda_state.threads_per_block_update, update_particle, 0, 0);
    cuda_state.grid_size_solver = n / cuda_state.threads_per_block_solver;
    cuda_state.grid_size_solver = n % cuda_state.threads_per_block_solver != 0 ? cuda_state.grid_size_solver + 1 : cuda_state.grid_size_solver;
    cuda_state.grid_size_solver = cuda_state.grid_size_solver == 0 ? 1 : cuda_state.grid_size_solver;
    cuda_state.grid_size_update = n / cuda_state.threads_per_block_update;
    cuda_state.grid_size_update = n % cuda_state.threads_per_block_update != 0 ? cuda_state.grid_size_update + 1 : cuda_state.grid_size_update;
    cuda_state.grid_size_update = cuda_state.grid_size_update == 0 ? 1 : cuda_state.grid_size_update;
    LOG(LOG_INFO, "Using %d blocks with %d threads = %d for solver\n", cuda_state.grid_size_solver, cuda_state.threads_per_block_solver, cuda_state.threads_per_block_solver * cuda_state.grid_size_solver);
    LOG(LOG_INFO, "Using %d blocks with %d threads = %d for update\n", cuda_state.grid_size_update, cuda_state.threads_per_block_update, cuda_state.threads_per_block_update * cuda_state.grid_size_update);
}

void free_cuda_tick(void)
{
    hipFree(device_particles);
}

void cuda_tick(Particle *p, volatile int *running, int n, double dt)
{
    BoundingBox b_box = compute_bounding_box(p, n);
    //LOG(LOG_INFO, "(%f, %f, %f) <-> (%f, %f, %f)\n", b_box.xmin, b_box.ymin, b_box.zmin, b_box.xmax, b_box.ymax, b_box.zmax);
    for (int i = 0; i < TIME_STEPS && *running; i++)
    {
        newtonianSolver<<<cuda_state.grid_size_solver, cuda_state.threads_per_block_solver>>>(device_particles, dt, n);
        update_particle<<<cuda_state.grid_size_update, cuda_state.threads_per_block_update>>>(device_particles, dt, n);
    }
    hipMemcpy(p, device_particles, n * sizeof(Particle), hipMemcpyDeviceToHost);
}

BoundingBox compute_bounding_box(Particle *p, int n)
{
    BoundingBox ret;
    ret.xmin = p[0].pos.x;
    ret.ymin = p[0].pos.y;
    ret.zmin = p[0].pos.z;
    ret.xmax = p[0].pos.x;
    ret.ymax = p[0].pos.y;
    ret.zmax = p[0].pos.z;
    for (int i = 0; i < n; i++)
    {
        ret.xmin = p[i].pos.x < ret.xmin ? p[i].pos.x : ret.xmin;
        ret.ymin = p[i].pos.y < ret.ymin ? p[i].pos.y : ret.ymin;
        ret.zmin = p[i].pos.z < ret.zmin ? p[i].pos.z : ret.zmin;
        ret.xmax = p[i].pos.x > ret.xmax ? p[i].pos.x : ret.xmax;
        ret.ymax = p[i].pos.y > ret.ymax ? p[i].pos.y : ret.ymax;
        ret.zmax = p[i].pos.z > ret.zmax ? p[i].pos.z : ret.zmax;
    }
    return ret;
}

__global__ void update_particle(Particle *particles, const double dt, int n)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n)
        return;
    Particle p = particles[tid];
    p.pos.x += p.vel.x * dt;
    p.pos.y += p.vel.y * dt;
    p.pos.z += p.vel.z * dt;
    particles[tid] = p;
}

#if USE_OCTREE

__global__ void newtonianSolver(Particle *particles, const double dt, int n)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n || tid == n - 1)
        return;

    Particle p = particles[tid];
    vec3_t accel = {0.0, 0.0, 0.0};

    for (int i = 0; i < n; i++)
    {
        if (i == tid)
            continue;

        Particle other = particles[i];

        vec3_t dist;
        dist.x = other.pos.x - p.pos.x;
        dist.y = other.pos.y - p.pos.y;
        dist.z = other.pos.z - p.pos.z;

        double r2 = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;
        double r = sqrt(r2);

        double f = other.mass / (r2 * r);

        accel.x += G * f * dist.x;
        accel.y += G * f * dist.y;
        accel.z += G * f * dist.z;
    }

    p.vel.x += accel.x * dt;
    p.vel.y += accel.y * dt;
    p.vel.z += accel.z * dt;

    particles[tid] = p;
}

#else

__global__ void newtonianSolver(Particle *particles, const double dt, int n)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n || tid == n - 1)
        return;

    Particle p = particles[tid];
    vec3_t accel = {0.0, 0.0, 0.0};

    for (int i = 0; i < n; i++)
    {
        if (i == tid)
            continue;

        Particle other = particles[i];

        vec3_t dist;
        dist.x = other.pos.x - p.pos.x;
        dist.y = other.pos.y - p.pos.y;
        dist.z = other.pos.z - p.pos.z;

        double r2 = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;
        double r = sqrt(r2);

        double f = other.mass / (r2 * r);

        accel.x += G * f * dist.x;
        accel.y += G * f * dist.y;
        accel.z += G * f * dist.z;
    }

    p.vel.x += accel.x * dt;
    p.vel.y += accel.y * dt;
    p.vel.z += accel.z * dt;

    particles[tid] = p;
}
#endif