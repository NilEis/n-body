#include "hip/hip_runtime.h"
#include "cuda_tick.h"

#define DELTA_TIME 0.001

__global__ void newtonianSolver(Particle *particles, const double dt);
__global__ void update_particle(Particle *particles, const double dt);

Particle *device_particles;

void init_cuda_tick(Particle *p)
{
    hipMalloc((void **)&device_particles, NUM_PARTICLES * sizeof(Particle));
    hipMemcpy(device_particles, p, NUM_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);
}

void free_cuda_tick(void)
{
    hipFree(device_particles);
}

void cuda_tick(Particle *p, volatile int *running)
{
    for (int i = 0; i < TIME_STEPS && *running; i++)
    {
        newtonianSolver<<<NUM_PARTICLES / CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK>>>(device_particles, DELTA_TIME);
        update_particle<<<NUM_PARTICLES / CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK>>>(device_particles, DELTA_TIME);
    }
    hipMemcpy(p, device_particles, NUM_PARTICLES * sizeof(Particle), hipMemcpyDeviceToHost);
}

__global__ void update_particle(Particle *particles, const double dt)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NUM_PARTICLES)
        return;
    Particle p = particles[tid];
    p.pos.x += p.vel.x * dt;
    p.pos.y += p.vel.y * dt;
    p.pos.z += p.vel.z * dt;
    particles[tid] = p;
}

__global__ void newtonianSolver(Particle *particles, const double dt)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= NUM_PARTICLES)
        return;

    Particle p = particles[tid];
    vec3_t accel = {0.0, 0.0, 0.0};

    for (int i = 0; i < NUM_PARTICLES; i++)
    {
        if (i == tid)
            continue;

        Particle other = particles[i];

        vec3_t dist;
        dist.x = other.pos.x - p.pos.x;
        dist.y = other.pos.y - p.pos.y;
        dist.z = other.pos.z - p.pos.z;

        double r2 = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;
        double r = sqrt(r2);

        double f = other.mass / (r2 * r);

        accel.x += f * dist.x;
        accel.y += f * dist.y;
        accel.z += f * dist.z;
    }

    p.vel.x += accel.x * dt;
    p.vel.y += accel.y * dt;
    p.vel.z += accel.z * dt;

    particles[tid] = p;
}