#include "hip/hip_runtime.h"
#include "cuda_tick.h"

#define G 0.06

static int threads_per_block_solver = 0;
static int threads_per_block_update = 0;
static int grid_size_solver = 0;
static int grid_size_update = 0;

__global__ void newtonianSolver(Particle *particles, const double dt, int n);
__global__ void update_particle(Particle *particles, const double dt, int n);

Particle *device_particles;

void init_cuda_tick(Particle *p, int n)
{
    hipMalloc((void **)&device_particles, n * sizeof(Particle));
    hipMemcpy(device_particles, p, n * sizeof(Particle), hipMemcpyHostToDevice);
    hipOccupancyMaxPotentialBlockSize(&grid_size_solver, &threads_per_block_solver, newtonianSolver, 0, 0);
    hipOccupancyMaxPotentialBlockSize(&grid_size_update, &threads_per_block_update, update_particle, 0, 0);
    grid_size_solver = n / threads_per_block_solver;
    grid_size_solver = n % threads_per_block_solver != 0 ? grid_size_solver + 1 : grid_size_solver;
    grid_size_solver = grid_size_solver == 0 ? 1 : grid_size_solver;
    grid_size_update = n / threads_per_block_update;
    grid_size_update = n % threads_per_block_update != 0 ? grid_size_update + 1 : grid_size_update;
    grid_size_update = grid_size_update == 0 ? 1 : grid_size_update;
    LOG(LOG_INFO, "Using %d blocks with %d threads = %d for solver\n", grid_size_solver, threads_per_block_solver, threads_per_block_solver * grid_size_solver);
    LOG(LOG_INFO, "Using %d blocks with %d threads = %d for update\n", grid_size_update, threads_per_block_update, threads_per_block_update * grid_size_update);
}

void free_cuda_tick(void)
{
    hipFree(device_particles);
}

void cuda_tick(Particle *p, volatile int *running, int n, double dt)
{
    for (int i = 0; i < TIME_STEPS && *running; i++)
    {
        newtonianSolver<<<grid_size_solver, threads_per_block_solver>>>(device_particles, dt, n);
        update_particle<<<grid_size_update, threads_per_block_update>>>(device_particles, dt, n);
    }
    hipMemcpy(p, device_particles, n * sizeof(Particle), hipMemcpyDeviceToHost);
}

__global__ void update_particle(Particle *particles, const double dt, int n)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n)
        return;
    Particle p = particles[tid];
    p.pos.x += p.vel.x * dt;
    p.pos.y += p.vel.y * dt;
    p.pos.z += p.vel.z * dt;
    particles[tid] = p;
}

__global__ void newtonianSolver(Particle *particles, const double dt, int n)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n || tid == n - 1)
        return;

    Particle p = particles[tid];
    vec3_t accel = {0.0, 0.0, 0.0};

    for (int i = 0; i < n; i++)
    {
        if (i == tid)
            continue;

        Particle other = particles[i];

        vec3_t dist;
        dist.x = other.pos.x - p.pos.x;
        dist.y = other.pos.y - p.pos.y;
        dist.z = other.pos.z - p.pos.z;

        double r2 = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z;
        double r = sqrt(r2);

        double f = other.mass / (r2 * r);

        accel.x += G * f * dist.x;
        accel.y += G * f * dist.y;
        accel.z += G * f * dist.z;
    }

    p.vel.x += accel.x * dt;
    p.vel.y += accel.y * dt;
    p.vel.z += accel.z * dt;

    particles[tid] = p;
}